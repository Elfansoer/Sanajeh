#include "hip/hip_runtime.h"
#include "sanajeh_device_code.h"

AllocatorHandle<AllocatorT>* allocator_handle;
__device__ AllocatorT* device_allocator;

__device__ Body::Body(float px, float py, float vx, float vy, float fx, float fy, float m) {
	this->pos_x = px;
	this->pos_y = py;
	this->vel_x = vx;
	this->vel_y = vy;
	this->force_x = fx;
	this->force_y = fy;
	this->mass = m;
}

__device__ Body::Body(int idx) {
	hiprandState rand_state;
	hiprand_init(kSeed, idx, 0, &rand_state);
	this->pos_x = (2.0 * hiprand_uniform(&rand_state)) - 1.0;
	this->pos_y = (2.0 * hiprand_uniform(&rand_state)) - 1.0;
	this->vel_x = 0.0;
	this->vel_y = 0.0;
	this->mass = ((hiprand_uniform(&rand_state) / 2.0) + 0.5) * kMaxMass;
	this->force_x = 0.0;
	this->force_y = 0.0;
}

__device__ void Body::compute_force() {
	this->force_x = 0.0;
	this->force_y = 0.0;
	device_allocator->template device_do<Body>(&Body::apply_force, this);
}

__device__ void Body::apply_force(Body* other) {
	if (other != this) {
		float dx = this->pos_x - other->pos_x;
		float dy = this->pos_x - other->pos_y;
		float dist = sqrt((dx * dx) + (dy * dy));
		float f = ((kGravityConstant * this->mass) * other->mass) / ((dist * dist) + kDampeningFactor);
		other->force_x += (f * dx) / dist;
		other->force_y += (f * dy) / dist;
	}
}

__device__ void Body::body_update() {
	this->vel_x += (this->force_x * kDt) / this->mass;
	this->vel_y += (this->force_y * kDt) / this->mass;
	this->pos_x += this->vel_x * kDt;
	this->pos_y += this->vel_y * kDt;
	if (this->pos_x < -1 || this->pos_x > 1) {
		this->vel_x = -this->vel_x;
	}
	if (this->pos_y < -1 || this->pos_y > 1) {
		this->vel_y = -this->vel_y;
	}
}

void Body::_do(void (*pf)(float, float, float, float, float, float, float)){
	pf(this->pos_x, this->pos_y, this->vel_x, this->vel_y, this->force_x, this->force_y, this->mass);
}

extern "C" int Body_do_all(void (*pf)(float, float, float, float, float, float, float)){
	allocator_handle->template device_do<Body>(&Body::_do, pf);
 	return 0;
}

extern "C" int Body_Body_compute_force(){
	allocator_handle->parallel_do<Body, &Body::compute_force>();
	return 0;
}

extern "C" int Body_Body_body_update(){
	allocator_handle->parallel_do<Body, &Body::body_update>();
	return 0;
}

extern "C" int parallel_new_Body(int object_num){
	allocator_handle->parallel_new<Body>(object_num);
	return 0;
}

extern "C" int AllocatorInitialize(){
	allocator_handle = new AllocatorHandle<AllocatorT>(/* unified_memory= */ true);
	AllocatorT* dev_ptr = allocator_handle->device_pointer();
	hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0, hipMemcpyHostToDevice);
	return 0;
}