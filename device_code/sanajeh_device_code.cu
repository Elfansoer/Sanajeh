#include "hip/hip_runtime.h"
#include "sanajeh_device_code.h"

AllocatorHandle<AllocatorT>* allocator_handle;
__device__ AllocatorT* device_allocator;

__device__ Body::Body(float px, float py, float vx, float vy, float fx, float fy, float m) {
	this->pos_x = px;
	this->pos_y = py;
	this->vel_x = vx;
	this->vel_y = vy;
	this->force_x = fx;
	this->force_y = fy;
	this->mass = m;
}

__device__ Body::Body(int idx) {
	hiprandState rand_state;
	hiprand_init(kSeed, idx, 0, &rand_state);
	this->pos_x = (2.0 * hiprand_uniform(&rand_state)) - 1.0;
	this->pos_y = (2.0 * hiprand_uniform(&rand_state)) - 1.0;
	this->vel_x = (hiprand_uniform(&rand_state) - 0.5) / 1000.0;
	this->vel_y = (hiprand_uniform(&rand_state) - 0.5) / 1000.0;
	this->mass = ((hiprand_uniform(&rand_state) / 2.0) + 0.5) * kMaxMass;
	this->force_x = 0.0;
	this->force_y = 0.0;
}

__device__ void Body::compute_force() {
	this->force_x = 0.0;
	this->force_y = 0.0;
	device_allocator->template device_do<Body>(&Body::apply_force, this);
}

__device__ void Body::apply_force(Body* other) {
	if (other != this) {
		float dx = this->pos_x - other->pos_x;
		float dy = this->pos_x - other->pos_y;
		float dist = sqrt((dx * dx) + (dy * dy));
		float f = ((kGravityConstant * this->mass) * other->mass) / ((dist * dist) + kDampeningFactor);
		other->force_x += (f * dx) / dist;
		other->force_y += (f * dy) / dist;
	}
}

__device__ void Body::body_update() {
	this->vel_x += (this->force_x * kDt) / this->mass;
	this->vel_y += (this->force_y * kDt) / this->mass;
	this->pos_x += this->vel_x * kDt;
	this->pos_y += this->vel_y * kDt;
	if (this->pos_x < -1 || this->pos_x > 1) {
		this->vel_x = -this->vel_x;
	}
	if (this->pos_y < -1 || this->pos_y > 1) {
		this->vel_y = -this->vel_y;
	}
}

void Body::_do(void (*pf)(float, float, float, float, float, float, float)){
	pf(this->pos_x, this->pos_y, this->vel_x, this->vel_y, this->force_x, this->force_y, this->mass);
}

extern "C" int Body_do_all(void (*pf)(float, float, float, float, float, float, float)){
	allocator_handle->template device_do<Body>(&Body::_do, pf);
 	return 0;
}

extern "C" int Body_Body_compute_force(){
	allocator_handle->parallel_do<Body, &Body::compute_force>();
	return 0;
}

extern "C" int Body_Body_body_update(){
	allocator_handle->parallel_do<Body, &Body::body_update>();
	return 0;
}

extern "C" int parallel_new_Body(int object_num){
	allocator_handle->parallel_new<Body>(object_num);
	return 0;
}

extern "C" int AllocatorInitialize(){
	allocator_handle = new AllocatorHandle<AllocatorT>(/* unified_memory= */ true);
	AllocatorT* dev_ptr = allocator_handle->device_pointer();
	hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0, hipMemcpyHostToDevice);
	return 0;
}

int main(int argc, char* argv[]) {

	AllocatorInitialize();
	parallel_new_Body(atoi(argv[1]));
	long unsigned int overall = 0;

	for (int i = 0; i < 100; ++i) {
		auto time_start = std::chrono::system_clock::now();
		Body_Body_compute_force();
		Body_Body_body_update();
		auto time_end = std::chrono::system_clock::now();
		auto elapsed = time_end - time_start;
		auto micros = std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
	}
	printf("%lu\n", overall/100);

	return 0;
}